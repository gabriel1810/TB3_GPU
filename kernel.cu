#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>

__global__ void vectorAdd(int* a, int* b, int* c, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    // Tamanho dos vetores, ajustar conforme memoria disponivel
    int n = 268435456;

    int *hostMatrizA, *hostMatrizB, *hostMatrizC, * hostMatrizD;

    int* gpuMatrizA, * gpuMatrizB, * gpuMatrizC;

    hostMatrizA = (int*)malloc(n * sizeof(int));
    hostMatrizB = (int*)malloc(n * sizeof(int));
    hostMatrizC = (int*)malloc(n * sizeof(int));
    hostMatrizD = (int*)malloc(n * sizeof(int));

    for (int i = 0; i < n; i++) {
        hostMatrizA[i] = i;
        hostMatrizB[i] = i;
        hostMatrizD[i] = hostMatrizA[i] + hostMatrizB[i];
    }

    // Alterar para 1 caso queira usar GPU
    int usarGPU = 0;

    if (usarGPU) {
        // Alocar memoria na GPU
        hipMalloc((void**)&gpuMatrizA, n * sizeof(int));
        hipMalloc((void**)&gpuMatrizB, n * sizeof(int));
        hipMalloc((void**)&gpuMatrizC, n * sizeof(int));

        // Copiando vetores do Host para a GPU
        hipMemcpy(gpuMatrizA, hostMatrizA, n * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(gpuMatrizB, hostMatrizB, n * sizeof(int), hipMemcpyHostToDevice);
        
        int minGrid;
        int tamBloco;

        // Calcula o numero de blocos a serem alocados na GPU
        hipOccupancyMaxPotentialBlockSize(&minGrid, &tamBloco, vectorAdd, 0, n);
        int tamGrid = (n + tamBloco - 1) / tamBloco;

        // Executar a soma na GPU
        vectorAdd << <tamGrid, tamBloco >> > (gpuMatrizA, gpuMatrizB, gpuMatrizC, n);

        // Copiar o vetor novamente para o Host
        hipMemcpy(hostMatrizC, gpuMatrizC, n * sizeof(int), hipMemcpyDeviceToHost);
    }
    else {
            // Soma na CPU
            for (int i = 0; i < n; i++) {
                hostMatrizC[i] = (hostMatrizA[i] + hostMatrizB[i]);
             }
    }

    
    printf("n = %d\n", n);

    //Verifica se o resultado está OK
    for (int i = 0; i < n; i++) {
        if (hostMatrizC[i] != hostMatrizD[i]) {
            printf("Erro na posicao %d\n", i);
        }
    }
    printf("Calculo OK");


    if (usarGPU) {
        hipFree(gpuMatrizA);
        hipFree(gpuMatrizB);
        hipFree(gpuMatrizC);
    }

    free(hostMatrizA);
    free(hostMatrizB);
    free(hostMatrizC);

    return 0;
}
